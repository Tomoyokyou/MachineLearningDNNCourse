#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>
#include "sigmoid.h"

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <cuda_memory_manager.h>
#include "parser.h"

#include <random>

using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

template <typename T>
void pushOne(device_matrix<T>& m) {
  device_matrix<T> tmp(~m);
  T* h_data = new T [m.size()+m.getCols()];
  hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(T), hipMemcpyDeviceToHost);
  tmp.resize(tmp.getRows(),tmp.getCols()+1);
  for(size_t t=0;t<tmp.getRows();++t)
  h_data[m.size()+t]=1;
  hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(T), hipMemcpyHostToDevice);
  m=~tmp;
  delete [] h_data;
}

int main(int argc,char** argv){

PARSER p;

default_random_engine generator;
normal_distribution<float> dis(0,0.1);
size_t dim=500;
float n=0.02;
srand(time(0));

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

//testing element-wise operation

mat C(8,2), D(8,2,2.5);
randomInit(C);
randomInit(D);

//testing sigmoid function

Sigmoid n1(5,5);
/*
C.resize(8,3);
randomInit(C);

printf("testing push one \n");
pushOne(C);
C.print();

printf("testing ext::sigmoid\n");
(ext::sigmoid(C)).print();
*/
n1.print();

printf("divide by a const num\n");
C.print();
printf("\n");
(C*n/(float)dim).print();
/*
A.resize(5,8);B.resize(5,8);
randomInit(A);randomInit(B);

C.resize(5,5);
randomInit(C);
gemm(A,B,C,(float)-1,(float)1,false,true);
printf("C=\n");
C.print();
*/
printf("sigmoid test\n");
(ext::sigmoid(C)).print();
/*
C.print();
printf("pushone test\n");
pushOne(C);
C.print();
*/
return 0;
}
