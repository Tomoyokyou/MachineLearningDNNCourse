#include <iostream>
#include <vector>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>
#include "sigmoid.h"

using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

int main(){

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

printf("A=\n");
A.print();
printf("B=\n");
B.print();

printf("A * B= \n"); (A*B).print();

//testing element-wise operation

mat C(8,1), D(8,1);
randomInit(C);
randomInit(D);

printf("C=\n");
C.print();
printf("D=\n");
D.print();

printf("C & D= \n"); (C&D).print();

//testing sigmoid function

Sigmoid n1(5,5);

//float**

float** _fptr=new float*[10];
for(size_t t=0;t<10;++t){
	_fptr[t]=new float[20];
}

for(size_t t=0;t<10;++t){
	for(size_t k=0;k<20;++k)
		_fptr[t][k]=t+100*k;
}

float* test=_fptr[2];

for(size_t t=0;t<20;++t){
	cout<<" "<<test[t];
	if(t+1%5==0)
		cout<<endl;
}
cout<<endl;

for(size_t t=0;t<10;++t)
	delete [] _fptr[t];
delete [] _fptr;

return 0;
}
