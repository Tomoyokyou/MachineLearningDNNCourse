#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <cuda_memory_manager.h>
#include "parser.h"
#include "transforms.h"
#include <random>

using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

template <typename T>
void pushOne(device_matrix<T>& m) {
  device_matrix<T> tmp(~m);
  T* h_data = new T [m.size()+m.getCols()];
  hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(T), hipMemcpyDeviceToHost);
  tmp.resize(tmp.getRows(),tmp.getCols()+1);
  for(size_t t=0;t<tmp.getRows();++t)
  h_data[m.size()+t]=1;
  hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(T), hipMemcpyHostToDevice);
  m=~tmp;
  delete [] h_data;
}

int main(int argc,char** argv){

PARSER p;


default_random_engine gen((unsigned)time(NULL));
size_t dim=500;
float n=0.02;
srand(time(0));

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

//testing element-wise operation

mat C(8,2), D(8,2,2.5);
randomInit(C);
randomInit(D);

//testing sigmoid function

printf("divide by a const num\n");
C.print();
printf("\n");
(C*n/(float)dim).print();

mat in(10,3);
randomInit(in);
Softmax s1(10,10);
mat out;
s1.forward(out,in,true);
cout<<"out"<<endl;
out.print();
mat bk;
s1.backPropagate(bk,out,0.02,0);
cout<<"bk="<<endl;
bk.print();

Sigmoid s2(10,10);
cout<<"testing sigmoid"<<endl;
s2.forward(out,in,true);
cout<<"out"<<endl;
out.print();
s2.backPropagate(bk,out,0.02,0);
cout<<"bk="<<endl;
bk.print();

cout<<"testing normal distribution"<<endl;
Sigmoid sss(5,6);
cout<<endl;
Sigmoid ss2(5,6);
cout<<endl;
Softmax s12(5,6);
cout<<endl;
return 0;
}
