#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <cuda_memory_manager.h>
#include "parser.h"
#include "transforms.h"

#include <random>

using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

template <typename T>
void pushOne(device_matrix<T>& m) {
  device_matrix<T> tmp(~m);
  T* h_data = new T [m.size()+m.getCols()];
  hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(T), hipMemcpyDeviceToHost);
  tmp.resize(tmp.getRows(),tmp.getCols()+1);
  for(size_t t=0;t<tmp.getRows();++t)
  h_data[m.size()+t]=1;
  hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(T), hipMemcpyHostToDevice);
  m=~tmp;
  delete [] h_data;
}

int main(int argc,char** argv){

PARSER p;

default_random_engine generator;
normal_distribution<float> dis(0,0.1);
size_t dim=500;
float n=0.02;
srand(time(0));

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

//testing element-wise operation

mat C(8,2), D(8,2,2.5);
randomInit(C);
randomInit(D);

//testing sigmoid function

printf("divide by a const num\n");
C.print();
printf("\n");
(C*n/(float)dim).print();

mat in(10,3);
randomInit(in);
cout<<"in"<<endl;
in.print();
Softmax s1(10,10);
mat out;
s1.forward(out,in,true);
cout<<"out"<<endl;
out.print();
 

return 0;
}
