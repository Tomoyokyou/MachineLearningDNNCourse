#include <iostream>
#include <vector>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>
#include "sigmoid.h"

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <cuda_memory_manager.h>

using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

template <typename T>
void pushOne(device_matrix<T>& m) {
  device_matrix<T> tmp(~m);
  T* h_data = new T [m.size()+m.getCols()];
  hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(T), hipMemcpyDeviceToHost);
  tmp.resize(tmp.getRows(),tmp.getCols()+1);
  for(size_t t=0;t<tmp.getRows();++t)
  h_data[m.size()+t]=1;
  hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(T), hipMemcpyHostToDevice);
  m=~tmp;
  delete [] h_data;
}

int main(int argc,char** argv){

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

printf("A=\n");
A.print();
printf("B=\n");
B.print();

printf("A * B= \n"); (A*B).print();

//testing element-wise operation

mat C(8,2), D(8,2,2.5);
randomInit(C);
randomInit(D);

printf("C=\n");
C.print();
printf("C*2=\n");
(C * 2).print();
printf("D=\n");
D.print();

printf("C & D= \n"); (C&D).print();

//testing sigmoid function

Sigmoid n1(5,5);

//float**

float** _fptr=new float*[10];
for(size_t t=0;t<10;++t){
	_fptr[t]=new float[20];
}

for(size_t t=0;t<10;++t){
	for(size_t k=0;k<20;++k)
		_fptr[t][k]=t+100*k;
}

float* test=_fptr[2];

for(size_t t=0;t<20;++t){
	cout<<" "<<test[t];
	if(t+1%5==0)
		cout<<endl;
}
cout<<endl;

for(size_t t=0;t<10;++t)
	delete [] _fptr[t];
delete [] _fptr;

C.resize(8,3);
printf("C=\n");
C.print();

printf("testing push one \n");
pushOne(C);
C.print();

printf("testing ext::sigmoid\n");
(ext::sigmoid(C)).print();

n1.print();
C.print();
(C-1).print();

A.resize(5,8);B.resize(8,5);
randomInit(A);randomInit(B);
C.resize(5,5);
randomInit(C);
printf("C=\n");
C.print();
printf("A*B=\n");
mat tem=A*B;
tem.print();
gemm(A,B,C,(float)1,(float)2,false,false);
printf("C=\n");
C.print();


return 0;
}
