#include "mynngen.h"
#include <device_matrix.h>
#include <device_arithmetic.h>
#include <device_math.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <string>
#include <vector>
#include <cassert>
#include <cstdlib>

typedef device_matrix<float> mat;

myNnGen gn(0,0.2);

void rand_init(mat& w,float range){
	float* h_data = new float[w.size()];
	for(size_t t=0;t<w.getRows()*(w.getCols()-1);++t)
		h_data[t]=2*range*rand()/(float)RAND_MAX - range;
	for(size_t t=0;t<w.getRows();++t)
		h_data[t+w.getRows()*(w.getCols()-1)]=0;
	CCE(hipMemcpy(w.getData(),h_data,w.size()* sizeof(float) , hipMemcpyHostToDevice));
	delete [] h_data;
}
void rand_norm(mat& w,myNnGen& ran){
	float* h_data = new float[w.size()];
	for(size_t t=0;t<w.getRows()*(w.getCols()-1);++t)
		h_data[t]=ran();
	for(size_t t=0;t<w.getRows();++t)
		h_data[t+w.getRows()*(w.getCols()-1)]=0;
	CCE(hipMemcpy(w.getData(),h_data,w.size()* sizeof(float) , hipMemcpyHostToDevice));
	delete [] h_data;
}

void pushOne(mat& in){
	mat tmp(~in);
	thrust::device_vector<float> dvec(tmp.size()+tmp.getRows(),1);
	thrust::device_ptr<float> mat_ptr(tmp.getData());
	thrust::copy(mat_ptr,mat_ptr+tmp.size(),dvec.begin());
	thrust::device_ptr<float> vec_ptr=dvec.data();
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	thrust::device_ptr<float> mat_ptr2(tmp.getData());
	thrust::copy(dvec.begin(),dvec.end(),mat_ptr2);
	in = ~tmp;
}

void getBias(mat& out,const mat& w){
	float* h_data=new float[w.getRows()];
	CCE(hipMemcpy(h_data,w.getData()+(w.getRows())*(w.getCols()-1),sizeof(float)*w.getRows(),hipMemcpyDeviceToHost));
	out.resize(w.getRows(),1);
	CCE(hipMemcpy(out.getData(),h_data,sizeof(float)*w.getRows(),hipMemcpyHostToDevice));
	delete [] h_data;
}

void replaceBias(mat& w,const mat& bias){
	assert(bias.getCols()==1);
	assert(w.getRows()==bias.size());
	CCE(hipMemcpy(w.getData()+w.getRows()*(w.getCols()-1),bias.getData(),sizeof(float)*w.getRows(),hipMemcpyDeviceToDevice));
}

void parseDim(string str,vector<size_t>& dim){
	size_t begin=str.find_first_not_of(' '),end;
	string hold;
	while(begin!=string::npos){
		end=str.find_first_of('-',begin);
		if(end==string::npos)
			hold=str.substr(begin);
		else
			hold=str.substr(begin,end-begin);
		if(!hold.empty())
			dim.push_back(atoi(hold.c_str()));
		begin=str.find_first_not_of('-',end);
	}
}

