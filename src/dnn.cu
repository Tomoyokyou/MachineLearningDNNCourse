#include "hip/hip_runtime.h"
#include "dnn.h"
#include "dataset.h"
#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <fstream>
#include <cmath>
#include <cassert>
#include <ctime>
#include <device_matrix.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>
#include "util.h"

#define MAX_EPOCH 10000000

using namespace std;

typedef device_matrix<float> mat;

float computeErrRate(const vector<size_t>& ans, const vector<size_t>& output);
void computeLabel(vector<size_t>& result,const mat& outputMat);

DNN::DNN():_pData(NULL), _learningRate(0.001),_momentum(0), _method(ALL){}
DNN::DNN(Dataset* pData, float learningRate,float momentum,float variance,Init init, const vector<size_t>& v, Method method):_pData(pData), _learningRate(learningRate),_momentum(momentum), _method(method){
	int numOfLayers = v.size();
	switch(init){
	case NORMAL:
		gn.reset(0,variance);
		for(int i = 0; i < numOfLayers-1; i++){
			Transforms* pTransform;
			if( i < numOfLayers-2 )
				pTransform = new Sigmoid(v.at(i), v.at(i+1), gn);
			else
				pTransform = new Softmax(v.at(i), v.at(i+1), gn);
			_transforms.push_back(pTransform);
		}
		break;
	case UNIFORM:
	case RBM:
	default:
		for(int i = 0; i < numOfLayers-1; i++){
			Transforms* pTransform;
			if( i < numOfLayers-2 )
				pTransform = new Sigmoid(v.at(i), v.at(i+1), variance);
			else
				pTransform = new Softmax(v.at(i), v.at(i+1), variance);
			_transforms.push_back(pTransform);
		}
		break;
	}
}
DNN::~DNN(){
	while(!_transforms.empty()){
		delete _transforms.back();
		_transforms.pop_back();
	}
}

void DNN::train(size_t batchSize, size_t maxEpoch = MAX_EPOCH, size_t trainSetNum = 10000, size_t validSetNum = 10000, float alpha = 0.98){
	clock_t rt1 = clock();

	mat trainSet;
	vector<size_t> trainLabel;
	mat validSet;
	vector<size_t> validLabel;
	size_t EinRise = 0;
	float Ein = 1;
	float pastEin = Ein;
	float minEin = Ein;
	float Eout = 1;
	float pastEout = Eout;
	float minEout = Eout;
	
	_pData->getTrainSet(trainSetNum, trainSet, trainLabel);
	_pData->getValidSet(validSetNum, validSet, validLabel);

	clock_t rt2 = clock();
	cout << "Get train/validate set:" << (rt2-rt1)/CLOCKS_PER_SEC << endl;
	
	size_t num = 0;
	for(; num < maxEpoch; num++){
		clock_t rt3 = clock();
		mat batchData;
		mat batchLabel;
		mat batchOutput;
		_pData->getBatch(batchSize, batchData, batchLabel);
		
		clock_t rt4 = clock();
		feedForward(batchOutput, batchData, true);
		//
		vector<size_t> debug;
		float ERR=1.0;
		predict(debug, trainSet);
		ERR= computeErrRate(trainLabel,debug);
		if(ERR==1.0){
			cout<<"iter"<<num<<" encounter 100\% error"<<endl;
				cerr<<"ERROR: program overflow..."<<endl;
				ofstream core("dnn.dump");
				for(size_t t=0;t<_transforms.size();++t)
					_transforms.at(t)->dump(core);
					core<<"\n last output:\n";
					batchOutput.print(core);
					core<<"\n";
					core<<"first gradient "<<endl;
					mat tempOut(batchOutput-batchLabel);
					tempOut.print(core);
					core<<endl;
			exit(1);
		}
		//

		clock_t rt5 = clock();
		mat lastDelta(batchOutput - batchLabel);
		backPropagate(lastDelta, _learningRate, _momentum); //momentum

		clock_t rt6 = clock();	
		
		if( num % 2000 == 0 )
			_learningRate *= alpha;

		if( num % 5000 == 1 ){

			clock_t rt7 = clock();
			vector<size_t> trainResult;
			vector<size_t> validResult;
			predict(trainResult, trainSet);
			predict(validResult, validSet);

			clock_t rt8 = clock();
			Ein = computeErrRate(trainLabel, trainResult);
			Eout = computeErrRate(validLabel, validResult);
			
			clock_t rt9 = clock();

			/*Print debug message here*/
			double duration = (rt9-rt3);
			//cout << "Per iteration: " << duration/CLOCKS_PER_SEC << " sec\n";
			//cout << "Get Batch time: " << (rt4-rt3)/duration << endl;
			//cout << "Feedforward: " << (rt5-rt4)/duration << endl;
			//cout << "Backpropagation: " << (rt6-rt5)/duration << endl;
			//cout << "Predict train/valid err: " << (rt8-rt7)/duration << endl;
			//cout << "Compute train/valid err: " << (rt9-rt8)/duration << endl;


			pastEin  = Ein;
			pastEout = Eout;
			if(minEin > Ein){
				minEin = Ein;
			}
			if(minEout > Eout){
				minEout = Eout;
				cout << "bestMdl: Error at: " << minEout << endl;  
				if(minEout < 0.5){
					ofstream ofs("best.mdl");
					if (ofs.is_open()){
						for(size_t i = 0; i < _transforms.size(); i++){
							(_transforms.at(i))->write(ofs);
						}
					}
					ofs.close();
				}
			}
			
			cout.precision(5);
			cout << "Validating error: " << Eout*100 << " %, Training error: " << Ein*100 << " %,  iterations:" << num-1 <<"\n";
		}
	}
	cout << "Finished training for " << num << " iterations.\n";
	cout << "bestMdl: Error at: " << minEout << endl;  
}

void DNN::predict(vector<size_t>& result, const mat& inputMat){
	mat outputMat(1, 1);
	feedForward(outputMat, inputMat, false);
	computeLabel(result, outputMat);
	/*  Transpose matrix print.
	for(size_t i = 0; i < outputMat.getRows(); i++){
		for(size_t j = 0; j < outputMat.getCols(); j++){
			cout << h_data[j*outputMat.getRows() + i] << " ";
		}
		cout << endl;
	}
	
	cout << endl;
	*/
	//delete [] h_data;
}

void DNN::setDataset(Dataset* pData){
	_pData = pData;
}
void DNN::setLearningRate(float learningRate){
	_learningRate = learningRate;
}
void DNN::setMomentum(float momentum){
	_momentum = momentum;
}

size_t DNN::getInputDimension(){
	return _transforms.front()->getInputDim();
}

size_t DNN::getOutputDimension(){
	return _transforms.back()->getOutputDim();
}

size_t DNN::getNumLayers(){
	return _transforms.size()+1;
}

void DNN::save(const string& fn){
	ofstream ofs(fn);
	if (ofs.is_open()){
		for(size_t i = 0; i < _transforms.size(); i++){
			(_transforms.at(i))->write(ofs);
		}
	}
	ofs.close();
}

bool DNN::load(const string& fn){
	ifstream ifs(fn);
	char buf[50000];
	if(!ifs){return false;}
	else{
		while(ifs.getline(buf, sizeof(buf)) != 0 ){
			string tempStr(buf);
			size_t found = tempStr.find_first_of(">");
			if(found !=std::string::npos ){
				size_t typeBegin = tempStr.find_first_of("<") + 1;
				string type = tempStr.substr(typeBegin, 7);
				stringstream ss(tempStr.substr(found+1));
				string rows, cols;
				size_t rowNum, colNum;
				ss >> rows >> cols;
				rowNum = stoi(rows);
				colNum = stoi(cols);
				size_t totalEle = rowNum * colNum;
				float* h_data = new float[totalEle];
				float* h_data_bias = new float[rowNum];
				for(size_t i = 0; i < rowNum; i++){
					if(ifs.getline(buf, sizeof(buf)) == 0){
						cerr << "Wrong file format!\n";
					}
					tempStr.assign(buf);
					stringstream ss1(tempStr);	
					for(size_t j = 0; j < colNum; j++){
						ss1 >> h_data[ j*rowNum + i ];
					}
				}
				ifs.getline(buf, sizeof(buf));
				ifs.getline(buf, sizeof(buf));
				tempStr.assign(buf);
				stringstream ss2(tempStr);
				float temp;
				for(size_t i = 0; i < rowNum; i++){
					ss2 >> h_data_bias[i];
				}
				mat weightMat(rowNum, colNum);
				mat biasMat(rowNum, 1);		
				hipMemcpy(weightMat.getData(), h_data, totalEle * sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(biasMat.getData(), h_data_bias, rowNum * sizeof(float), hipMemcpyHostToDevice);
				
				Transforms* pTransform;
				if(type == "sigmoid")
					pTransform = new Sigmoid(weightMat, biasMat);
				else if(type == "softmax")
					pTransform = new Softmax(weightMat, biasMat);
				else{
					cerr << "Undefined activation function! \" " << type << " \"\n";
					exit(1);
				}
				_transforms.push_back(pTransform);
				delete [] h_data;
				delete [] h_data_bias;
			}
		}
	}
	ifs.close();
	return true;
}

void DNN::feedForward(mat& outputMat, const mat& inputMat, bool train){
	mat tempInputMat = inputMat;
	for(size_t i = 0; i < _transforms.size(); i++){
		(_transforms.at(i))->forward(outputMat, tempInputMat, train);
		tempInputMat = outputMat;
	}
}

//The delta of last layer = _sigoutdiff & grad(errorFunc())
void DNN::backPropagate(const mat& deltaMat, float learningRate, float momentum){
	mat tempMat = deltaMat;
	mat errorMat;
	for(int i = _transforms.size()-1; i >= 0; i--){
		(_transforms.at(i))->backPropagate(errorMat, tempMat, learningRate, momentum);
		tempMat = errorMat;
	}
}

//Helper Functions
/*
mat posteriorProb2Label(const mat& prob) {
	assert(prob.getCols() > 1);
	size_t rows = prob.getRows(), cols = prob.getCols();

	hmat h_prob(prob);
  	hmat h_labels(1, cols);

  	for (size_t j=0; j<cols; ++j) {
		float max = -1e10;
    	size_t maxIdx = 0;

    	for (size_t i=0; i<rows; ++i) {
      		if (h_prob(i, j) > max) {
				max = h_prob(i, j);
				maxIdx = i;
      		}
    	}
    	h_labels[j] = maxIdx;
  	}
  return h_labels;
}
*/

size_t countDifference(const mat& m1, const mat& m2) {
	assert(m1.size() == m2.size());
	
	size_t L = m1.size();
  	thrust::device_ptr<float> ptr1(m1.getData());
 	thrust::device_ptr<float> ptr2(m2.getData());

  	size_t nDiff = thrust::inner_product(ptr1, ptr1 + L, ptr2, 0.0, thrust::plus<float>(), thrust::not_equal_to<float>());
  	return nDiff;
}

void computeLabel(vector<size_t>& result,const mat& outputMat){

	//int data[6] = {1, 0, 2, 2, 1, 3};
	//int result = thrust::reduce(thrust::host, data, data + 6, -1, thrust::maximum<int>()); // result == 3
	//thrust::device_vector<float>::iterator iter = thrust::max_element(d_vec.begin(), d_vec.end());

	//unsigned int position = iter - d_vec.begin();
	//float max_val = *iter;
	
	size_t inputDim = outputMat.getRows();
	size_t featureNum = outputMat.getCols();
	thrust::device_ptr<float> d_ptr = thrust::device_pointer_cast(outputMat.getData());
	thrust::host_vector<float> h_vec(d_ptr, d_ptr + inputDim*featureNum);
	for(size_t j = 0; j < outputMat.getCols(); j++){
		thrust::host_vector<float>::iterator iter = thrust::max_element(h_vec.begin() + j*inputDim, h_vec.begin() + (j+1)*inputDim);
		unsigned int position = iter - h_vec.begin() - j*inputDim;
		result.push_back(position);
	}

	/*
	float* h_data = new float [outputMat.size()];
	hipMemcpy(h_data ,outputMat.getData(), outputMat.size() * sizeof(float), hipMemcpyDeviceToHost);

	for(size_t j = 0; j < outputMat.getCols(); j++){
		float tempMax = h_data[j*outputMat.getRows()];
		size_t idx = 0;		
		for(size_t i = 0; i < outputMat.getRows(); i++){
			if(tempMax < h_data[j*outputMat.getRows() + i]){
				tempMax = h_data[j*outputMat.getRows() + i];
				idx = i;
			}
		}
		result.push_back(idx);
	}
	delete [] h_data;
	*/
}

float computeErrRate(const vector<size_t>& ans, const vector<size_t>& output){
	assert(ans.size() == output.size());
	size_t accCount = 0;
	for(size_t i = 0; i < ans.size(); i++){
		if(ans.at(i) == output.at(i)){
			accCount++;
		}
	}
	return 1.0-(float)accCount/(float)ans.size();
}
