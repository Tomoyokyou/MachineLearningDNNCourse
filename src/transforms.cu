#include "hip/hip_runtime.h"
#include "transforms.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <cstdlib>

#include <device_matrix.h>
#include <device_arithmetic.h>
#include <device_math.h>

<<<<<<< HEAD
//#include <random>

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

=======
#include <random>
>>>>>>> FETCH_HEAD

using namespace std;
using namespace ext;

<<<<<<< HEAD
typedef device_matrix<float> mat;
/////////////helper functions//////////////////////
=======
//helper functions
>>>>>>> FETCH_HEAD
void rand_init(mat& w){
	float* h_data = new float[w.size()];
	for(size_t t=0;t<w.getRows()*(w.getCols()-1);++t)
		h_data[t]=2*rand()/RAND_MAX-1;
	for(size_t t=0;t<w.getRows();++t)
		h_data[t+w.getRows()*(w.getCols()-1)]=0;
	CCE(hipMemcpy(w.getData(),h_data,w.size()* sizeof(float) , hipMemcpyHostToDevice));
	delete [] h_data;
}
void pushOne(mat& in){
	mat tmp(~in);
	float* h_data=new float[in.getRows()*(in.getCols()+1)];
<<<<<<< HEAD
	CCE(hipMemcpy(h_data,tmp.getData(),tmp.size()*sizeof(float),hipMemcpyDeviceToHost));
	for(size_t t=0;t<tmp.getRows();++t)
		h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	CCE(hipMemcpy(tmp.getData(),h_data,tmp.size()*sizeof(float),hipMemcpyHostToDevice));
	in = ~tmp;
	delete [] h_data;
}

template<typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
	T C;

	__host__ __device__
	linear_index_to_row_index(T C) : C(C) {}
	
	__host__ __device__
	T operator()(T i)
	{
			return i/C;
	}
};

void substractMaxPerRow(mat& x);
mat getRowMax(mat& C);
__global__ void substract_max_per_row(float* const A,float* const rmax, unsigned int rows , unsigned int cols);

void substractMaxPerRow(mat& x) {
	mat rmax = getRowMax(x);

	const int N = 32;
	dim3 grid;
	grid.x = (unsigned int) ceil((float) x.getCols() / N );
	grid.y = (unsigned int) ceil((float) x.getRows() / N );
	dim3 threads(N,N);

	substract_max_per_row<<<grid, threads>>>(x.getData(),rmax.getData() , x.getRows(),x.getCols());
	CCE(hipDeviceSynchronize());
}


__global__ void substract_max_per_row(float* const A, float * const rmax, unsigned int rows,unsigned int cols){
	int x = blockIdx.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= cols|| y>= rows)
			return;
	A[x * rows +y] -= rmax[y];
}

mat getRowMax(mat& C)
{
	mat rmax(C.getRows(),1);
	mat At = ~C;
	thrust::device_vector<float>row_indices(C.getRows());
	thrust::device_vector<float>row_results(C.getRows());
	thrust::reduce_by_key
	(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C.getCols())),
	 thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C.getCols())) +C.size(),thrust::device_ptr<float>(At.getData()),row_indices.begin(),
	 thrust::device_ptr<float>(rmax.getData()),thrust::equal_to<float>(),thrust::maximum<float>());
	
	return rmax;
}
//////////////////////////////////////////////
=======
	hipMemcpy(h_data,tmp.getData();tmp.size()*sizeof(float),hipMemcpyDeviceToHost);
	for(size_t t=0;t<tmp.getRows();++t)
		h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	hipMemcpy(tmp.getData(),h_data,tmp.size()*sizeof(float),hipMemcpyHostToDevice);
	in = ~tmp;
}
>>>>>>> FETCH_HEAD
/*
void rand_norm(float var,mat&){}
*/
///
<<<<<<< HEAD
=======
typedef device_matrix<float> mat;
>>>>>>> FETCH_HEAD


///////TRANSFORMS/////////////

Transforms::Transforms(const Transforms& t):_w(t._w),_i(t._i),_pw(t._pw){}

Transforms::Transforms(const mat& w):_w(w){
	_pw.resize(w.getRows(),w.getCols(),0);
}

Transforms::Transforms(size_t inputdim,size_t outputdim){
	_w.resize(outputdim,inputdim);
	rand_init(_w);
	_pw.resize(outputdim,inputdim,0);
}

size_t Transforms::getInputDim()const{
	return _w.getCols();
}
size_t Transforms::getOutputDim()const{
	return _w.getRows();
}

void Transforms::write(ofstream& out){
<<<<<<< HEAD
	float* h_data = new float[_w.size()];
	CCE(hipMemcpy( h_data, _w.getData(), _w.size() * sizeof(float), hipMemcpyDeviceToHost));
    out<<"<sigmoid> "<<_w.getRows()<<" "<<_w.getCols() - 1<<endl;
    for(size_t i=0;i<_w.getRows();++i){
    for(size_t j=0;j<_w.getCols()-1;++j){
                out<<" "<<h_data[_w.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_w.getRows()<<endl;
    for(size_t t=0;t<_w.getRows();++t)
                out<<" "<<h_data[_w.getRows()*(_w.getCols()-1)+t];
=======
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));
    out<<"<sigmoid> "<<_weight.getRows()<<" "<<_weight.getCols() - 1<<endl;
    for(size_t i=0;i<_weight.getRows();++i){
    for(size_t j=0;j<_weight.getCols()-1;++j){
                out<<" "<<h_data[_weight.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_weight.getRows()<<endl;
    for(size_t t=0;t<_weight.getRows();++t)
                out<<" "<<h_data[_weight.getRows()*(_weight.getCols()-1)+t];
>>>>>>> FETCH_HEAD
	out << endl;
	delete [] h_data;
}
void Transforms::print(){
	cout<<"Weight matrix: last column is bias"<<endl;
	_w.print();
	cout<<endl;
}
///////////////////////////////
/////////SIGMOID///////////////

<<<<<<< HEAD
Sigmoid::Sigmoid(const Sigmoid& s): Transforms(s){
}
Sigmoid::Sigmoid(const mat& w, const mat& bias): Transforms(w){
}
Sigmoid::Sigmoid(size_t inputdim,size_t outputdim): Transforms(inputdim,outputdim){
=======
Sigmoid::Sigmoid(const Sigmoid& s):_w(s._w),_i(s._i),_pw(s._pw){}
Sigmoid::Sigmoid(const mat& w):_w(w){
	_pw.resize(w.getRows(),w.getCols(),0);
}
Sigmoid::Sigmoid(size_t inputdim,size_t outputdim){
	_w.resize(outputdim,inputdim);
	rand_init(_w);
	_pw.resize(outputdim,inputdim,0);
>>>>>>> FETCH_HEAD
}
void Sigmoid::forward(mat& out,const mat& in,bool train){
	mat _inp(in);
	pushOne(_inp);
	out=sigmoid(_w*_inp);
	if(train){
		_i=in;
	}
}
<<<<<<< HEAD
void Sigmoid::backPropagate(mat& out,const mat& delta, float rate,float momentum){
	assert( (delta.getRows()==_w.getRows()) && (delta.getCols()==_i.getCols()) );
	mat withoutBias(_w.getRows(),_w.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_w.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat _tmp( ~withoutBias * delta);
	mat one(_i.getRows(),_i.getCols(),1);
	out = _i & (one-_i) & _tmp;   // this part need tesing
	// update weight
	mat _inp(_i);
	pushOne(_inp);
	_pw= delta * ~_inp + _pw * momentum;
	//_w -= _pw * rate;
	//NOTE: below are the case without momentum
	//rate=rate/(float)_input.getCols();
	gemm(delta,_inp,_w,(float)-1.0*rate,(float)1.0,false,true);
}
/*
=======
void Sigmoid::backPropagetion(mat& out,const mat& delta, float rate,float momentum){
	assert( (delta.getRows()==_weight.getRows()) && (delta.getCols()==_input.getCols()) );
	mat withoutBias(_weight.getRows(),_weight.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_weight.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat _tmp( ~withoutBias * delta);
	mat one(_input.getRows(),_input.getCols(),1);
	out = _input & (one-_input) & _tmp;   // this part need tesing
	// update weight
	mat _inp(_input);
	pushOne(_inp);
	_pw= delta * ~_inp;
	//NOTE: below are the case without momentum
	//rate=rate/(float)_input.getCols();
	gemm(delta,_inp,_weight,(float)-1.0*rate,(float)1.0,false,true);
}

>>>>>>> FETCH_HEAD
Sigmoid& Sigmoid::operator=(const Sigmoid& s){
	_w=s._w;
	_i=s._i;
	_pw=s._pw;
		return *this
}
<<<<<<< HEAD
*/
///////////////////////////////
///////////SOFTMAX/////////////

Softmax::Softmax(const Softmax& s): Transforms(s){
}
Softmax::Softmax(const mat& w, const mat& bias):Transforms(w){
}
Softmax::Softmax(size_t inputdim,size_t outputdim): Transforms(inputdim,outputdim){
}
void Softmax::forward(mat& out,const mat& in,bool train){
	mat inp=in;
	pushOne(inp);
	mat z=~(_w * inp);
	substractMaxPerRow(z);
	z=~z; // transpose to column vectors
	mat p(z.getRows(), z.getCols());
	
	thrust::device_ptr<float> zPtr(z.getData());
	thrust::device_ptr<float> pPtr(p.getData());
	thrust::transform(zPtr, zPtr + z.size(),zPtr, func::exp<float>());

	mat sumOfProb =  (mat(p.getRows(), p.getRows(),0) += 1) * p;
	out.resize(_w.getRows(),in.getCols());
	thrust::device_ptr<float> outptr(out.getData());
	thrust::device_ptr<float> sPtr(sumOfProb.getData());
	thrust::transform(pPtr,pPtr+p.size(), sPtr,outptr,thrust::divides<float>());
=======
///////////////////////////////
///////////SOFTMAX/////////////

Softmax::Softmax(const Softmax s):_w(s._w),_i(s._i),_pw(s._pw){
}
Softmax::Softmax(const mat& w):_w(w){
	_pw.resize(w.getRows(),w.getCols(),0);
}
Softmax::Softmax(size_t inputdim,size_t outputdim){
	_w.resize(outputdim,inputdim);
	rand_init(_w);
	_pw.resize(outputdim,inputdim);
}
//TODO
void Softmax::forward(mat& out,const mat& in,bool train){
	mat inp=in;
	pushOne(inp);
	mat z=_w * inp;
	//substractMax(z);
>>>>>>> FETCH_HEAD

	if(train){
		_i=in;
	}
}

<<<<<<< HEAD
void Softmax::backPropagate(mat& out,const mat& delta,float rate, float momentum){
	mat inp(_i);
	pushOne(inp);	
	_pw=delta * ~inp + _pw * momentum;
	//_w-= _pw * rate;
	//NOTE: eq. below haven't include momentum yet.
	gemm(delta,inp,_w,(float)-1.0*rate,(float)1.0,false,true);
	
}

=======
>>>>>>> FETCH_HEAD
///////////////////////////////
