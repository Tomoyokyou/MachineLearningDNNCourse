#include "transforms.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <cstdlib>

#include <device_matrix.h>
#include <device_arithmetic.h>
#include <device_math.h>

#include <random>

using namespace std;
using namespace ext;

//helper functions
void rand_init(mat& w){
	float* h_data = new float[w.size()];
	for(size_t t=0;t<w.getRows()*(w.getCols()-1);++t)
		h_data[t]=2*rand()/RAND_MAX-1;
	for(size_t t=0;t<w.getRows();++t)
		h_data[t+w.getRows()*(w.getCols()-1)]=0;
	CCE(hipMemcpy(w.getData(),h_data,w.size()* sizeof(float) , hipMemcpyHostToDevice));
	delete [] h_data;
}
void pushOne(mat& in){
	mat tmp(~in);
	float* h_data=new float[in.getRows()*(in.getCols()+1)];
	hipMemcpy(h_data,tmp.getData();tmp.size()*sizeof(float),hipMemcpyDeviceToHost);
	for(size_t t=0;t<tmp.getRows();++t)
		h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	hipMemcpy(tmp.getData(),h_data,tmp.size()*sizeof(float),hipMemcpyHostToDevice);
	in = ~tmp;
}
/*
void rand_norm(float var,mat&){}
*/
///
typedef device_matrix<float> mat;


///////TRANSFORMS/////////////

Transforms::Transforms(const Transforms& t):_w(t._w),_i(t._i),_pw(t._pw){}

Transforms::Transforms(const mat& w):_w(w){
	_pw.resize(w.getRows(),w.getCols(),0);
}

Transforms::Transforms(size_t inputdim,size_t outputdim){
	_w.resize(outputdim,inputdim);
	rand_init(_w);
	_pw.resize(outputdim,inputdim,0);
}

size_t Transforms::getInputDim()const{
	return _w.getCols();
}
size_t Transforms::getOutputDim()const{
	return _w.getRows();
}

void Transforms::write(ofstream& out){
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));
    out<<"<sigmoid> "<<_weight.getRows()<<" "<<_weight.getCols() - 1<<endl;
    for(size_t i=0;i<_weight.getRows();++i){
    for(size_t j=0;j<_weight.getCols()-1;++j){
                out<<" "<<h_data[_weight.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_weight.getRows()<<endl;
    for(size_t t=0;t<_weight.getRows();++t)
                out<<" "<<h_data[_weight.getRows()*(_weight.getCols()-1)+t];
	out << endl;
	delete [] h_data;
}
void Transforms::print(){
	cout<<"Weight matrix: last column is bias"<<endl;
	_w.print();
	cout<<endl;
}
///////////////////////////////
/////////SIGMOID///////////////

Sigmoid::Sigmoid(const Sigmoid& s):_w(s._w),_i(s._i),_pw(s._pw){}
Sigmoid::Sigmoid(const mat& w):_w(w){
	_pw.resize(w.getRows(),w.getCols(),0);
}
Sigmoid::Sigmoid(size_t inputdim,size_t outputdim){
	_w.resize(outputdim,inputdim);
	rand_init(_w);
	_pw.resize(outputdim,inputdim,0);
}
void Sigmoid::forward(mat& out,const mat& in,bool train){
	mat _inp(in);
	pushOne(_inp);
	out=sigmoid(_w*_inp);
	if(train){
		_i=in;
	}
}
void Sigmoid::backPropagetion(mat& out,const mat& delta, float rate,float momentum){
	assert( (delta.getRows()==_weight.getRows()) && (delta.getCols()==_input.getCols()) );
	mat withoutBias(_weight.getRows(),_weight.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_weight.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat _tmp( ~withoutBias * delta);
	mat one(_input.getRows(),_input.getCols(),1);
	out = _input & (one-_input) & _tmp;   // this part need tesing
	// update weight
	mat _inp(_input);
	pushOne(_inp);
	_pw= delta * ~_inp;
	//NOTE: below are the case without momentum
	//rate=rate/(float)_input.getCols();
	gemm(delta,_inp,_weight,(float)-1.0*rate,(float)1.0,false,true);
}

Sigmoid& Sigmoid::operator=(const Sigmoid& s){
	_w=s._w;
	_i=s._i;
	_pw=s._pw;
		return *this
}
///////////////////////////////
///////////SOFTMAX/////////////

Softmax::Softmax(const Softmax s):_w(s._w),_i(s._i),_pw(s._pw){
}
Softmax::Softmax(const mat& w):_w(w){
	_pw.resize(w.getRows(),w.getCols(),0);
}
Softmax::Softmax(size_t inputdim,size_t outputdim){
	_w.resize(outputdim,inputdim);
	rand_init(_w);
	_pw.resize(outputdim,inputdim);
}
//TODO
void Softmax::forward(mat& out,const mat& in,bool train){
	mat inp=in;
	pushOne(inp);
	mat z=_w * inp;
	//substractMax(z);

	if(train){
		_i=in;
	}
}

///////////////////////////////
