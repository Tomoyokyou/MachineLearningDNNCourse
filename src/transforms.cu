#include "hip/hip_runtime.h"
#include "transforms.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <cstdlib>
#include <string>

#include <device_matrix.h>
#include <device_arithmetic.h>
#include <device_math.h>

#include "util.h"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


using namespace std;
using namespace ext;

typedef device_matrix<float> mat;
/////////////helper functions//////////////////////
void rand_init(mat& w){
	float* h_data = new float[w.size()];
	for(size_t t=0;t<w.getRows()*(w.getCols()-1);++t)
		h_data[t]=2*rand()/(float)RAND_MAX - 1;
	for(size_t t=0;t<w.getRows();++t)
		h_data[t+w.getRows()*(w.getCols()-1)]=0;
	CCE(hipMemcpy(w.getData(),h_data,w.size()* sizeof(float) , hipMemcpyHostToDevice));
	delete [] h_data;
}
void rand_norm(mat& w){
	float* h_data = new float[w.size()];
	for(size_t t=0;t<w.getRows()*(w.getCols()-1);++t)
		h_data[t]=gn();
	for(size_t t=0;t<w.getRows();++t)
		h_data[t+w.getRows()*(w.getCols()-1)]=0;
	CCE(hipMemcpy(w.getData(),h_data,w.size()* sizeof(float) , hipMemcpyHostToDevice));
	delete [] h_data;
}
void pushOne(mat& in){
	mat tmp(~in);
	float* h_data=new float[(in.getRows()+1)*in.getCols()];
	CCE(hipMemcpy(h_data,tmp.getData(),tmp.size()*sizeof(float),hipMemcpyDeviceToHost));
	for(size_t t=0;t<tmp.getRows();++t)
		h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	CCE(hipMemcpy(tmp.getData(),h_data,tmp.size()*sizeof(float),hipMemcpyHostToDevice));
	in = ~tmp;
	delete [] h_data;
}

template<typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
	T C;

	__host__ __device__
	linear_index_to_row_index(T C) : C(C) {}
	
	__host__ __device__
	T operator()(T i)
	{
			return i/C;
	}
};

void substractMaxPerRow(mat& x);
mat getRowMax(mat& C);
__global__ void substract_max_per_row(float* const A,float* const rmax, unsigned int rows , unsigned int cols);

void substractMaxPerRow(mat& x) {
	mat rmax = getRowMax(x);

	const int N = 32;
	dim3 grid;
	grid.x = (unsigned int) ceil((float) x.getCols() / N );
	grid.y = (unsigned int) ceil((float) x.getRows() / N );
	dim3 threads(N,N);

	substract_max_per_row<<<grid, threads>>>(x.getData(),rmax.getData() , x.getRows(),x.getCols());
	CCE(hipDeviceSynchronize());
}


__global__ void substract_max_per_row(float* const A, float * const rmax, unsigned int rows,unsigned int cols){
	int x = blockIdx.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= cols|| y>= rows)
			return;
	A[x * rows +y] -= rmax[y];
}

mat getRowMax(mat& C)
{
	mat rmax(C.getRows(),1);
	mat At = ~C;
	thrust::device_vector<float>row_indices(C.getRows());
	thrust::device_vector<float>row_results(C.getRows());
	thrust::reduce_by_key
	(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C.getCols())),
	 thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C.getCols())) +C.size(),thrust::device_ptr<float>(At.getData()),row_indices.begin(),
	 thrust::device_ptr<float>(rmax.getData()),thrust::equal_to<float>(),thrust::maximum<float>());
	
	return rmax;
}
//////////////////////////////////////////////
///////TRANSFORMS/////////////

Transforms::Transforms(const Transforms& t):_w(t._w),_i(t._i),_pw(t._pw){}

Transforms::Transforms(const mat& w,const mat& b){
	assert(b.getRows()==1 || b.getCols()==1);
	size_t r=b.getRows(),c=b.getCols();
	if(r==1){r=c;c=1;}
	assert(w.getRows()==r);
	float* h_data=new float[w.size()+b.size()];
	float* b_data=new float[b.size()];
	CCE(hipMemcpy(h_data,w.getData(),w.size() *sizeof(float) ,hipMemcpyDeviceToHost));
	CCE(hipMemcpy(b_data,w.getData(),b.size() *sizeof(float) ,hipMemcpyDeviceToHost));
	for(size_t t=0;t<b.size();++t)
			h_data[w.size()+t]=b_data[t];
	_w.resize(w.getRows(),w.getCols()+1);
	CCE(hipMemcpy(_w.getData(),h_data,(w.size()+b.size()) * sizeof(float), hipMemcpyHostToDevice));
	delete [] b_data;
	delete [] h_data;
	_pw.resize(_w.getRows(),_w.getCols(),0);
}

Transforms::Transforms(size_t inputdim,size_t outputdim){
	_w.resize(outputdim,inputdim+1);
	//rand_norm(_w);  // default variance = 0.2 , to change varance head to include/util.h
	rand_init(_w); // uniform distribution
	_w/=sqrt((float)inputdim);
	_pw.resize(outputdim,inputdim+1,0);
}

size_t Transforms::getInputDim()const{
	return _w.getCols();
}
size_t Transforms::getOutputDim()const{
	return _w.getRows();
}

void Transforms::print(ofstream& out){
	float* h_data = new float[_w.size()];
	CCE(hipMemcpy( h_data, _w.getData(), _w.size() * sizeof(float), hipMemcpyDeviceToHost));
    for(size_t i=0;i<_w.getRows();++i){
    for(size_t j=0;j<_w.getCols()-1;++j){
                out<<" "<<h_data[_w.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_w.getRows()<<endl;
    for(size_t t=0;t<_w.getRows();++t)
                out<<" "<<h_data[_w.getRows()*(_w.getCols()-1)+t];
	out << endl;
	delete [] h_data;
}
///////////////////////////////
/////////SIGMOID///////////////

Sigmoid::Sigmoid(const Sigmoid& s): Transforms(s){
}
Sigmoid::Sigmoid(const mat& w, const mat& bias): Transforms(w,bias){
}
Sigmoid::Sigmoid(size_t inputdim,size_t outputdim): Transforms(inputdim,outputdim){
}
void Sigmoid::forward(mat& out,const mat& in,bool train){
	mat _inp(in);
	pushOne(_inp);
	out=sigmoid(_w * _inp);
	if(train){
		_i=in;
	}
}
void Sigmoid::backPropagate(mat& out,const mat& delta, float rate,float momentum){
	assert( (delta.getRows()==_w.getRows()) && (delta.getCols()==_i.getCols()) );
	mat withoutBias(_w.getRows(),_w.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_w.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat one(_i.getRows(),_i.getCols(),1);
	out = _i & (one-_i) & (~withoutBias * delta);   // this part need tesing
	// update weight
	mat _inp(_i);
	pushOne(_inp);
	_pw= delta * ~_inp + _pw * momentum;
	//NOTE: below are the case without momentum
	rate/=(float)_i.getCols();
	_w -= _pw * rate;
	//gemm(delta,_inp,_w,(float)-1.0*rate,(float)1.0,false,true);
}
void Sigmoid::write(ofstream& out){
	out<<"<sigmoid> "<<_w.getRows()<<" "<<_w.getCols()-1<<endl;
	print(out);
}

///////////////////////////////
///////////SOFTMAX/////////////

Softmax::Softmax(const Softmax& s): Transforms(s){
}
Softmax::Softmax(const mat& w, const mat& bias):Transforms(w,bias){
}
Softmax::Softmax(size_t inputdim,size_t outputdim): Transforms(inputdim,outputdim){
}
void Softmax::forward(mat& out,const mat& in,bool train){
	mat inp=in;
	pushOne(inp);
	mat z=~(_w * inp);
	substractMaxPerRow(z);
	z=~z; // transpose to column vectors
	mat p(z.getRows(), z.getCols());
	
	thrust::device_ptr<float> zPtr(z.getData());
	thrust::device_ptr<float> pPtr(p.getData());
	thrust::transform(zPtr, zPtr + z.size(),pPtr, func::exp<float>());

	mat sumOfProb =  (mat(p.getRows(), p.getRows(),0) += 1) * p;
	out.resize(_w.getRows(),in.getCols());
	thrust::device_ptr<float> outptr(out.getData());
	thrust::device_ptr<float> sPtr(sumOfProb.getData());
	thrust::transform(pPtr,pPtr+p.size(), sPtr,outptr,thrust::divides<float>());

	if(train){
		_i=in;
	}
}

void Softmax::backPropagate(mat& out,const mat& delta,float rate, float momentum){
	assert( (delta.getRows()==_w.getRows()) && (delta.getCols()==_i.getCols()) );
	mat withoutBias(_w.getRows(),_w.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_w.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat one(_i.getRows(),_i.getCols(),1);
	out = _i & (one-_i) & (~withoutBias * delta);   // this part need tesing
	//update weight
	mat inp(_i);
	pushOne(inp);	
	_pw=delta * ~inp + _pw * momentum;
	//NOTE: eq. below haven't include momentum yet.
	rate/=(float)_i.getCols();
	_w-= _pw * rate;
	//gemm(delta,inp,_w,(float)-1.0*rate,(float)1.0,false,true);
	
}
void Softmax::write(ofstream& out){
	out<<"<softmax> "<<_w.getRows()<<" "<<_w.getCols()-1<<endl;
	print(out);
}
///////////////////////////////
