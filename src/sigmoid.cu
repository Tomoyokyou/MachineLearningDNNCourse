#include "hip/hip_runtime.h"
#include "sigmoid.h"
#include <device_matrix.h>
#include <vector>
#include <fstream>
#include <cassert>
#include <cmath>
#include <ctime>
#include <cstdlib>
// nvcc compiler
#include <device_arithmetic.h>
#include <device_math.h>
#include <random>

using namespace std;

typedef device_matrix<float> mat;

Sigmoid::Sigmoid(){
	_weight.resize(1,2,0);
	rand_init();
}
Sigmoid::Sigmoid(const Sigmoid& s){
	_weight=s._weight;
	//NOTE::copy constructor won't copy input!
}
Sigmoid::Sigmoid(const mat& wpart,const mat& bias){
	size_t r=bias.getRows(), c=bias.getCols();
	assert(r==1 || c==1);
	if(c==1){
		c=r;r=1; //swap
	}
	assert(wpart.getRows()==c);
	float* h_data= new float[wpart.size()+bias.size()];
	float* b_data= new float[bias.size()];
	CCE(hipMemcpy(h_data,wpart.getData(), wpart.size() * sizeof(float),hipMemcpyDeviceToHost));
	CCE(hipMemcpy(b_data,bias.getData(),bias.size() * sizeof(float),hipMemcpyDeviceToHost));
	for(size_t t=0;t<bias.size();++t)
		h_data[t+wpart.size()]=b_data[t];

	_weight.resize(wpart.getRows(),wpart.getCols()+1);
	CCE(hipMemcpy(_weight.getData(),h_data,_weight.size() * sizeof(float),hipMemcpyHostToDevice));

	delete [] h_data;
	delete [] b_data;
}
Sigmoid::Sigmoid(const mat& w){
	_weight=w;
}
Sigmoid::Sigmoid(size_t out_dim, size_t inp_dim){
	_weight.resize(out_dim,inp_dim+1);  // +1 for bias
	//rand_init(); // uniform -0.5 ~ 0.5
	init_norm(0.1); // variance=0.1
	_weight=_weight/(float)sqrt(inp_dim);
	_prediff.resize(out_dim,inp_dim+1,0);
}

Sigmoid::~Sigmoid(){
}

Sigmoid& Sigmoid::operator = (const Sigmoid& sig){
	_weight=sig._weight;
	_input=sig._input;
	_prediff=sig._prediff;
	return *this;
}

void Sigmoid::forward(mat& out, const mat& in, bool train){
	mat _inp = mat(in);
	pushOne(_inp);
	//fill with 1 for computation simplicity
	out = ext::sigmoid( (_weight * _inp));
	if(train){
		_input = in;
	}
}

// assume error pass through var "delta"
void Sigmoid::backPropagate(mat& out, const mat& delta, float rate, float momentum){
	assert( (delta.getRows()==_weight.getRows()) && (delta.getCols()==_input.getCols()) );
	mat withoutBias(_weight.getRows(),_weight.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_weight.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat _tmp( (~withoutBias) * delta);
	mat one(_input.getRows(),_input.getCols(),1);
	out = _input & (one-_input) & _tmp;   // this part need tesing
	// update weight
	mat _inp(_input);
	pushOne(_inp);
	rate=rate/(float)_input.getCols();
	mat nw = ( (delta * ~inp) * -rate) + (_prediff * momentum);
	_weight += nw;
	_prediff = nw;
	//nw.print();    //next weight change;
	//gemm(delta,_inp,_weight,(float)-1.0*rate,(float)1.0,false,true);
}

void Sigmoid::getSigDiff(mat& delta,const mat& error){
	assert( (error.getRows()==_weight.getRows()) && (error.getCols()==_input.getCols()) );
	mat one(_weight.getRows(),_input.getCols(),1);
	mat _inp(_input);
	pushOne(_inp);
	delta = (_weight * _inp);
	mat sig=ext::sigmoid(delta);
	delta = (sig) & (one-sig) & error;
}

void Sigmoid::write(ofstream& out){
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));
    out<<"<sigmoid> "<<_weight.getRows()<<" "<<_weight.getCols() - 1<<endl;
    for(size_t i=0;i<_weight.getRows();++i){
    for(size_t j=0;j<_weight.getCols()-1;++j){
                out<<" "<<h_data[_weight.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_weight.getRows()<<endl;
    for(size_t t=0;t<_weight.getRows();++t)
                out<<" "<<h_data[_weight.getRows()*(_weight.getCols()-1)+t];
	out << endl;
	delete [] h_data;
}

void Sigmoid::print(FILE* fid, int precision, char delimiter){
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));

	char format[16];
	sprintf(format,"%c%%.%de",delimiter,(precision>0)? precision :0);
	
	fprintf(fid,"<sigmoid> %d %d \n",_weight.getRows() ,_weight.getCols()-1); // <sigmoid> outputDimension inputDimension
	for(size_t i=0;i<_weight.getRows();++i){
		for(size_t j=0;j<_weight.getCols()-1;++j)
			fprintf(fid,format,h_data[j*_weight.getRows()+i]);
		fprintf(fid,"\n");
	}
	
	fprintf(fid,"<bias> %d \n",_weight.getRows()); // <bias> output dimensions
	for(size_t t=0;t<_weight.getRows();++t)
		fprintf(fid,format,h_data[_weight.getRows() * (_weight.getCols()-1) + t]);
	fprintf(fid,"\n");
	
	delete [] h_data;
}
size_t Sigmoid::getInputDim(){
	return _weight.getCols()-1;
}
size_t Sigmoid::getOutputDim(){
	return _weight.getRows();
}
void Sigmoid::rand_init(){
	size_t _s=_weight.size();
	float* h_data = new float [_s];
	for (size_t i=0; i<_s; ++i)
		h_data[i]=(rand() / (float) RAND_MAX) -0.5;
	CCE(hipMemcpy(_weight.getData(), h_data, _weight.size() * sizeof(float), hipMemcpyHostToDevice));
	delete [] h_data;
}

void Sigmoid::init_norm(float var){
	default_random_engine eng;
	normal_distribution<float> dis(0,var);
	size_t s=_weight.size();
	float* h_data =new float [s];
	for(size_t t=0;t<s;++t)
			h_data[t]=dis(eng);
	CCE(hipMemcpy(_weight.getData(),h_data,_weight.size() * sizeof(float),hipMemcpyHostToDevice));
}

void Sigmoid::pushOne(mat& input){
	device_matrix<float> tmp(~input);
    float* h_data = new float [input.size()+input.getCols()];
	CCE(hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(float), hipMemcpyDeviceToHost));
    for(size_t t=0;t<tmp.getRows();++t)
	h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	CCE(hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(float), hipMemcpyHostToDevice));
    input=~tmp;
	delete [] h_data;
}

