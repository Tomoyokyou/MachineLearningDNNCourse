#include "sigmoid.h"
#include <device_matrix.h>
#include <vector>
#include <fstream>
#include <cassert>
// nvcc compiler
#include <device_arithmetic.h>
#include <device_math.h>

using namespace std;

typedef device_matrix<float> mat;

Sigmoid::Sigmoid(){
	_weight.resize(1,2);
	_sigout.resize(2,1);
	_input.resize(1,1);
	_weight.fillwith(0);
}
Sigmoid::Sigmoid(const mat& w){
	_weight=w;
	_sigout.resize(_weight.getRows(),1);
	_input.resize(_weight.getCols()-1,1);
}
Sigmoid::Sigmoid(size_t out_dim, size_t inp_dim){
	_weight.resize(out_dim,inp_dim+1);  // +1 for bias
	_sigout.resize(out_dim,1);
	_input.resize(inp_dim,1);
	rand_init();
}

Sigmoid::~Sigmoid(){
}

void Sigmoid::forward(mat& out, const mat& in, bool train){
	//assume in is a vector
	mat _inp = mat(in);
	pushOne(_inp);
	//fill with 1 for computation simplicity
	out = ext::sigmoid( (_weight * _inp));
	//if in training mode 
	if(train){
		_input = in;
		_sigout = _weight * (_inp);	
	}
}

// assume error pass through var "delta"
void Sigmoid::backPropagate(mat& out, const mat& delta, float rate){
	mat _tmp( (~_weight) * delta);
	mat one(_tmp.getRows(),_tmp.getCols(),1);
	out= _tmp & _sigout & (one-_sigout) ;   // this part need tesing
	// update weight
	mat _inp(_input);
	pushOne(_inp);
	gemm(out,_inp,_weight,-rate,(float)1.0,false,true);

}

void Sigmoid::write(FILE* out){
	_weight.print(out,4,' ');
}

void Sigmoid::print(FILE* fid, int precision, char delimiter){
	float* h_data = new float[_weight.size()];
	hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost);

	char format[16];
	sprintf(format,"%c%%.%de",delimiter,(precision>0)? precision :0);
	fprint(fid,"<sigmoid> %d %d \n",_weight.getRows() ,_weight.getCols()) // <sigmoid> outputDimension inputDimension
	for(size_t i=0;i<_weight.getRows();++i){
		for(size_t j=0;j_weight.getCols()-1;++j)
			fprintf(fid,format,h_data[j*_weight.getRows()+i]);
		fprintf(fid,"\n");
	}
	fprintf(fid,"<bias> %d",_weight.getRows()); // <bias> output dimensions
	for(size_t t=0;t<_weight.getRows();++t)
		fprintf(fid,format,h_data[_weight.getRows() * (_weight.getCols()-1) + t]);
	fprintf(fid,"\n");
	delete [] h_data;
}
size_t Sigmoid::getInputDim(){
	return _weight.getCols()-1;
}
size_t Sigmoid::getOutputDim(){
	return _weight.getRows();
}
void Sigmoid::rand_init(){
	size_t _s=_weight.size();
	float* h_data = new float [_s];
	for (size_t i=0; i<_s; ++i)
		h_data[i]=rand() / (float) RAND_MAX;
	hipMemcpy(_weight.getData(), h_data, _weight.size() * sizeof(float), hipMemcpyHostToDevice);
	delete [] h_data;
}

void Sigmoid::pushOne(mat& input){
	device_matrix<float> tmp(~input);
    float* h_data = new float [input.size()+input.getCols()];
	hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(float), hipMemcpyDeviceToHost);
    for(size_t t=0;t<tmp.getRows();++t)
	h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(float), hipMemcpyHostToDevice);
    input=~tmp;
	delete [] h_data;
}

// element-wise operation

