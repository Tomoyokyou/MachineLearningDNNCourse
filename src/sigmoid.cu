#include "hip/hip_runtime.h"
#include "sigmoid.h"
#include <device_matrix.h>
#include <vector>
#include <fstream>
#include <cassert>
#include <cmath>
#include <ctime>
#include <cstdlib>
// nvcc compiler
#include <device_arithmetic.h>
#include <device_math.h>

using namespace std;

typedef device_matrix<float> mat;

Sigmoid::Sigmoid(){
	_weight.resize(1,2,0);
	_input.resize(1,1,0);
	_prediff.resize(1,1,0);
}
Sigmoid::Sigmoid(const Sigmoid& s){
	_weight=s._weight;
	_input.resize(1,1,0);
	_prediff.resize(1,1,0);
	//NOTE::copy constructor won't copy input!
}
Sigmoid::Sigmoid(const mat& wpart,const mat& bias){
	size_t r=bias.getRows(), c=bias.getCols();
	assert(r==1 || c==1);
	if(c==1){
		c=r;r=1; //swap
	}
	assert(wpart.getRows()==c);
	float* h_data= new float[wpart.size()+bias.size()];
	float* b_data= new float[bias.size()];
	CCE(hipMemcpy(h_data,wpart.getData(), wpart.size() * sizeof(float),hipMemcpyDeviceToHost));
	CCE(hipMemcpy(b_data,bias.getData(),bias.size() * sizeof(float),hipMemcpyDeviceToHost));
	for(size_t t=0;t<bias.size();++t)
		h_data[t+wpart.size()]=b_data[t];

	_weight.resize(wpart.getRows(),wpart.getCols()+1);
	CCE(hipMemcpy(_weight.getData(),h_data,_weight.size() * sizeof(float),hipMemcpyHostToDevice));

	delete [] h_data;
	delete [] b_data;
	_prediff.resize(1,1,0);
	_input.resize(1,1,0);
}
Sigmoid::Sigmoid(const mat& w){
	_weight=w;
	_input.resize(_weight.getCols()-1,1,0);
	_prediff.resize(1,1,0);
}
Sigmoid::Sigmoid(size_t out_dim, size_t inp_dim){
	_weight.resize(out_dim,inp_dim+1);  // +1 for bias
	rand_init();
	_input.resize(1,1,0);
	_prediff.resize(1,1,0);
	//_weight/=sqrt(inp_dim);
}

Sigmoid::~Sigmoid(){
}

Sigmoid& Sigmoid::operator = (const Sigmoid& sig){
	_weight=sig._weight;
	_input=sig._input;
	_prediff=sig._prediff;
	return *this;
}

void Sigmoid::forward(mat& out, const mat& in, bool train){
	mat _inp = mat(in);
	pushOne(_inp);
	//fill with 1 for computation simplicity
	out = ext::sigmoid( (_weight * _inp));
	if(train){
		_input = in;
	}
}

// assume error pass through var "delta"
void Sigmoid::backPropagate(mat& out, const mat& delta, float rate, float momentum){
	assert( (delta.getRows()==_weight.getRows()) && (delta.getCols()==_input.getCols()) );
	mat withoutBias(_weight.getRows(),_weight.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_weight.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat _tmp( (~withoutBias) * delta);
	mat one(_input.getRows(),_input.getCols(),1);
	mat diff= (_input) & (one-_input);
	out = diff & _tmp;   // this part need tesing
	// update weight
	mat _inp(_input);
	pushOne(_inp);
/*
	if(( _prediff.size()==_weight.size()) &&( momentum!= 0 ))
	_prediff = delta * ~_input + _prediff * momentum ;
	else
	_prediff = delta * ~_input;
	_weight = _weight -  _prediff * rate;
*/
	//_weight = _weight - prediff * (rate/(float)(_weight.getCols()-1);
	//NOTE: below are the case without momentum
	gemm(delta,_inp,_weight,(float)-1.0*rate,(float)1.0,false,true);
	//gemm(delta,_inp,_weight,(float)-1.0*rate/(float)_input.getCols(),(float)1.0,false,true);
}

void Sigmoid::getSigDiff(mat& delta,const mat& error){
	assert( (error.getRows()==_weight.getRows()) && (error.getCols()==_input.getCols()) );
	mat one(_weight.getRows(),_input.getCols(),1);
	mat _inp(_input);
	pushOne(_inp);
	delta = (_weight * _inp);
	mat sig=ext::sigmoid(delta);
	delta = (sig) & (one-sig) & error;
}

void Sigmoid::write(ofstream& out){
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));
    out<<"<sigmoid> "<<_weight.getRows()<<" "<<_weight.getCols()<<endl;
    for(size_t i=0;i<_weight.getRows();++i){
    for(size_t j=0;j<_weight.getCols()-1;++j){
                out<<" "<<h_data[_weight.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_weight.getRows()<<endl;
    for(size_t t=0;t<_weight.getRows();++t)
                out<<" "<<h_data[_weight.getRows()*(_weight.getCols()-1)+t];
	delete [] h_data;
}

void Sigmoid::print(FILE* fid, int precision, char delimiter){
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));

	char format[16];
	sprintf(format,"%c%%.%de",delimiter,(precision>0)? precision :0);
	
	fprintf(fid,"<sigmoid> %d %d \n",_weight.getRows() ,_weight.getCols()); // <sigmoid> outputDimension inputDimension
	for(size_t i=0;i<_weight.getRows();++i){
		for(size_t j=0;j<_weight.getCols()-1;++j)
			fprintf(fid,format,h_data[j*_weight.getRows()+i]);
		fprintf(fid,"\n");
	}
	
	fprintf(fid,"<bias> %d \n",_weight.getRows()); // <bias> output dimensions
	for(size_t t=0;t<_weight.getRows();++t)
		fprintf(fid,format,h_data[_weight.getRows() * (_weight.getCols()-1) + t]);
	fprintf(fid,"\n");
	
	delete [] h_data;
}
size_t Sigmoid::getInputDim(){
	return _weight.getCols()-1;
}
size_t Sigmoid::getOutputDim(){
	return _weight.getRows();
}
void Sigmoid::rand_init(){
	size_t _s=_weight.size();
	float* h_data = new float [_s];
	for (size_t i=0; i<_s; ++i)
		h_data[i]=(rand() / (float) RAND_MAX) -0.5;
	CCE(hipMemcpy(_weight.getData(), h_data, _weight.size() * sizeof(float), hipMemcpyHostToDevice));
	delete [] h_data;
}

void Sigmoid::pushOne(mat& input){
	device_matrix<float> tmp(~input);
    float* h_data = new float [input.size()+input.getCols()];
	CCE(hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(float), hipMemcpyDeviceToHost));
    for(size_t t=0;t<tmp.getRows();++t)
	h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	CCE(hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(float), hipMemcpyHostToDevice));
    input=~tmp;
	delete [] h_data;
}

