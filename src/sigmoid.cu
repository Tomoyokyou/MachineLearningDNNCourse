#include "sigmoid.h"
#include <device_matrix.h>
#include <vector>
#include <fstream>
#include <cassert>
// nvcc compiler
#include <device_arithmetic.h>
#include <device_math.h>

using namespace std;
using namespace ext;

typedef device_matrix<float> mat;

Sigmoid::Sigmoid(){
	_weight.resize(1,2);
	_sigout.resize(2,1);
	_input.resize(1,1);
	_weight->fillwith(0);
}
Sigmoid::Sigmoid(const mat& w){
	_weight=w;
	_sigout.resize(_weight.getRows(),1);
	_input.resize(_weight.getCols()-1,1);
}
Sigmoid::Sigmoid(size_t row, size_t col){
	_weight.resize(row,col+1);  // +1 for bias
	_sigout.resize(row,1);
	_input.resize(col,1);
	rand_init();
}
Sigmoid::~Sigmoid(){
}

void Sigmoid::forward(mat& out, const mat& in, bool train){
	//assume in is a vector
	mat* _inp = new mat(in);
	_inp->resize(in.getRows()+1,in.getCols());
	float* h_data=_inp->getData();
	h_data[in.getRows()]=1;
	//fill with 1 for computation simplicity
	out = sigmoid( _weight * (*_inp));
	//if in training mode 
	if(train){
		_input = in;
		_sigout = _weight * (*_inp);	
	}
	delete _inp;
}

// assume error pass through var "delta"
Sigmoid::backPropagate(mat& out, const mat& delta, float rate){
	mat _tmp( (~_weight) * delta);
	out= _tmp & _sigout & (1-_sigout) ;   // this part need tesing
	
	// update weight
	mat _inp(_input);
	_inp.resize(_input.getRows()+1,1);
	float* h_data = _inp.getData();
	h_data[_input.getRows()]=1;
	gemm(out,_inp,_weight,-rate,1.0,false,true);

}

void Sigmoid::print(ofstream& out){
}

size_t Sigmoid::getInputDim(){
	return _weight.getCols()-1;
}
size_t Sigmoid::getOutputDim(){
	return _weight.getRows();
}
void Sigmoid::rand_init(){
	size_t _s=_weight.size();
	float* h_data = new float [_s];
	for (size_t i=0; i<_s; ++i)
		h_data[i]=rand() / (float) RAND_MAX;
	hipMemcpy(_weight.getData(), h_data, _weight.size() * sizeof(float), hipMemcpyHostToDevice);
	delete [] h_data;
}

// element-wise operation

