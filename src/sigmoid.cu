#include "sigmoid.h"
#include <device_matrix.h>
#include <vector>
#include <fstream>
#include <cassert>
// nvcc compiler
#include <device_arithmetic.h>
#include <device_math.h>

using namespace std;

typedef device_matrix<float> mat;

Sigmoid::Sigmoid(){
	_weight.resize(1,2);
	_sigout.resize(2,1);
	_input.resize(1,1);
	_weight.fillwith(0);
}
Sigmoid::Sigmoid(const mat& w){
	_weight=w;
	_sigout.resize(_weight.getRows(),1);
	_input.resize(_weight.getCols()-1,1);
}
Sigmoid::Sigmoid(size_t out_dim, size_t inp_dim){
	_weight.resize(out_dim,inp_dim+1);  // +1 for bias
	_sigout.resize(out_dim,1);
	_input.resize(inp_dim,1);
	rand_init();
}

Sigmoid::~Sigmoid(){
}

void Sigmoid::forward(mat& out, const mat& in, bool train){
	//assume in is a vector
	mat _inp = mat(in);
	_inp.resize(in.getRows()+1,in.getCols());
	float* h_data=_inp.getData();
	h_data[in.getRows()]=1;
	//fill with 1 for computation simplicity
	out = ext::sigmoid( (_weight * _inp));
	//if in training mode 
	if(train){
		_input = in;
		_sigout = _weight * (_inp);	
	}
}

// assume error pass through var "delta"
void Sigmoid::backPropagate(mat& out, const mat& delta, float rate){
	mat _tmp( (~_weight) * delta);
	mat one(_tmp.getRows(),_tmp.getCols(),1);
	out= _tmp & _sigout & (one-_sigout) ;   // this part need tesing
	// update weight
	mat _inp(_input);
	_inp.resize(_input.getRows()+1,1);
	float* h_data = _inp.getData();
	h_data[_input.getRows()]=1;
	gemm(out,_inp,_weight,-rate,(float)1.0,false,true);

}

void Sigmoid::write(ostream& out){

}

void Sigmoid::print(){
	_weight.print();
}
size_t Sigmoid::getInputDim(){
	return _weight.getCols()-1;
}
size_t Sigmoid::getOutputDim(){
	return _weight.getRows();
}
void Sigmoid::rand_init(){
	size_t _s=_weight.size();
	float* h_data = new float [_s];
	for (size_t i=0; i<_s; ++i)
		h_data[i]=rand() / (float) RAND_MAX;
	hipMemcpy(_weight.getData(), h_data, _weight.size() * sizeof(float), hipMemcpyHostToDevice);
	delete [] h_data;
}

// element-wise operation

