#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>

using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

void pushOne(mat& in){
	mat tmp(~in);
	thrust::device_vector<float> dvec(tmp.size()+tmp.getRows());
	thrust::device_ptr<float> mat_ptr(tmp.getData());
	thrust::copy(mat_ptr,mat_ptr+tmp.size(),dvec.begin());
	thrust::device_ptr<float> vec_ptr=dvec.data();
	thrust::fill(vec_ptr+tmp.size()+1,vec_ptr+tmp.size()+1+tmp.getRows(),1);
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	thrust::device_ptr<float> mat_ptr2(tmp.getData());
	thrust::copy(dvec.begin(),dvec.end(),mat_ptr2);
	//CCE(hipMemcpy(tmp.getData(),dvec.data(),tmp.size()*sizeof(float),hipMemcpyDeviceToDevice));
	in = ~tmp;
}

template<typename T>
struct linear_index_to_col_index : public thrust::unary_function<T,T>
{
	T C;

	__host__ __device__
	linear_index_to_col_index(T C) : C(C) {}
	
	__host__ __device__
	T operator()(T i)
	{
			return i/C;
	}
};
void substractMaxPerCol(mat& x);
mat getColMax(mat& C);
__global__ void substract_max_per_col(float* const A,float* const rmax, unsigned int rows , unsigned int cols);

void substractMaxPerCol(mat& x) {
	mat rmax = getColMax(x);

	const int N = 32;
	dim3 grid;
	grid.x = (unsigned int) ceil((float) x.getCols() / N );
	grid.y = (unsigned int) ceil((float) x.getRows() / N );
	dim3 threads(N,N);

	substract_max_per_col<<<grid, threads>>>(x.getData(),rmax.getData() , x.getRows(),x.getCols());
	CCE(hipDeviceSynchronize());
}


__global__ void substract_max_per_col(float* const A, float * const rmax, unsigned int rows,unsigned int cols){
	int x = blockIdx.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= cols|| y>= rows)
			return;
	A[x * rows +y] -= rmax[x];
}
mat getColMax(mat& C)
{
	mat rmax(C.getCols(),1);
	thrust::device_vector<float>row_indices(C.getCols());
	thrust::device_vector<float>row_results(C.getCols());
	thrust::reduce_by_key
	(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_col_index<int>(C.getRows())),
	 thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_col_index<int>(C.getRows())) +C.size(),thrust::device_ptr<float>(C.getData()),row_indices.begin(),
	 thrust::device_ptr<float>(rmax.getData()),thrust::equal_to<float>(),thrust::maximum<float>());
	
	return rmax;
}
int main(){

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

//testing element-wise operation

mat C(8,1), D(8,1);
randomInit(C);
randomInit(D);

printf("C=\n");
C.print();
printf("D=\n");
D.print();

printf("C & D= \n"); ((C) & (D)).print();

cout<<"row max"<<endl;
A.print();
cout<<endl;
mat out=getColMax(A);
cout<<"result"<<endl;
out.print();
substractMaxPerCol(A);
cout<<endl;
A.print();
cout<<endl;
return 0;
}
